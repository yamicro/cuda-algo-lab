#include <iostream>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <hip/hip_runtime.h>
#include "bench_cuda/benchmark_cuda.h"
#include "cuda/add.cu"
#include "cuda/histgram.cu"
#include "cuda/sigmod.cu"
#include "cuda/relu.cu"
#include "cuda/elu.cu"
#include "cuda/gelu.cu"
#include "cuda/swish.cu"





float add_cuda(pybind11::array_t<float> a, pybind11::array_t<float> b, pybind11::array_t<float> c) {
    auto buf_a = a.unchecked<1>();
    auto buf_b = b.unchecked<1>();
    auto buf_c = c.mutable_unchecked<1>();
    int n = buf_a.size();

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, n * sizeof(float));
    hipMalloc(&d_b, n * sizeof(float));

    hipMemcpy(d_a, buf_a.data(0), n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, buf_b.data(0), n * sizeof(float), hipMemcpyHostToDevice);

    float elapsed = benchmark_kernel([&]() {
        add_kernel<<<(n+255)/256, 256>>>(d_a, d_b, d_c, n);
    }, 3, 10);


    hipMemcpy(buf_c.mutable_data(0), d_c, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return elapsed;
}


float histogram_cuda(pybind11::array_t<int> a, pybind11::array_t<int> y) {
    auto buf_a = a.unchecked<1>();
    auto buf_y = y.mutable_unchecked<1>();
    int N = buf_a.size();

    int *d_a, *d_y;
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_y, y.size() * sizeof(int));

    hipMemcpy(d_a, buf_a.data(0), N * sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(d_y, buf_y.data(0), buf_y.size(0) * sizeof(int), hipMemcpyHostToDevice);
    float elapsed = benchmark_kernel([&]() {
        histogram_kernel<<<(N+255)/256, 256>>>(d_a, d_y, N);
    }, 3, 10);


    hipMemcpy(buf_y.mutable_data(0), d_y, buf_y.size() * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_y);

    return elapsed;
}

float sigmoid_cuda(pybind11::array_t<float> x, pybind11::array_t<float> y) {
    auto buf_x = x.unchecked<1>();
    auto buf_y = y.mutable_unchecked<1>();
    int N = buf_x.size();

    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    hipMemcpy(d_x, buf_x.data(0), N * sizeof(float), hipMemcpyHostToDevice);

    float elapsed = benchmark_kernel([&]() {
        sigmoid_kernel<<<(N+255)/256, 256>>>(d_x, d_y, N);
    }, 3, 10);

    hipMemcpy(buf_y.mutable_data(0), d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

    return elapsed;
}

float relu_cuda(pybind11::array_t<float> x, pybind11::array_t<float> y) {
    auto buf_x = x.unchecked<1>();
    auto buf_y = y.mutable_unchecked<1>();
    int N = buf_x.size();

    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    hipMemcpy(d_x, buf_x.data(0), N * sizeof(float), hipMemcpyHostToDevice);

    // benchmark 包裹 kernel 启动
    float elapsed = benchmark_kernel([&]() {
        relu_kernel<<<(N+255)/256, 256>>>(d_x, d_y, N);
    },3, 10);

    hipMemcpy(buf_y.mutable_data(0), d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

    return elapsed;
}

float elu_cuda(pybind11::array_t<float> x, pybind11::array_t<float> y) {
    auto buf_x = x.unchecked<1>();
    auto buf_y = y.mutable_unchecked<1>();
    int N = buf_x.size();

    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    hipMemcpy(d_x, buf_x.data(0), N * sizeof(float), hipMemcpyHostToDevice);

	float elapsed = benchmark_kernel([&]() {
        relu_kernel<<<(N+255)/256, 256>>>(d_x, d_y, N);
    },3, 10);

    hipMemcpy(buf_y.mutable_data(0), d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    return elapsed;
}

float gelu_cuda(pybind11::array_t<float> x, pybind11::array_t<float> y) {
    auto buf_x = x.unchecked<1>();
    auto buf_y = y.mutable_unchecked<1>();
    int N = buf_x.size();

    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    hipMemcpy(d_x, buf_x.data(0), N * sizeof(float), hipMemcpyHostToDevice);

    float elapsed = benchmark_kernel([&]() {
    	gelu_kernel<<<(N + 255) / 256, 256>>>(d_x, d_y, N);
    },3, 10);

    hipMemcpy(buf_y.mutable_data(0), d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    return elapsed;
}

float swish_cuda(pybind11::array_t<float> x, pybind11::array_t<float> y) {
    auto buf_x = x.unchecked<1>();
    auto buf_y = y.mutable_unchecked<1>();
    int N = buf_x.size();

    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    hipMemcpy(d_x, buf_x.data(0), N * sizeof(float), hipMemcpyHostToDevice);

    float elapsed = benchmark_kernel([&]() {
    	swish_f32_kernel<<<(N + 255) / 256, 256>>>(d_x, d_y, N);
    },3, 10);

    hipMemcpy(buf_y.mutable_data(0), d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    return elapsed;
}

PYBIND11_MODULE(binding, m) {
    m.def("add_cuda", &add_cuda, "CUDA add two arrays");
    m.def("histogram_cuda", &histogram_cuda, "CUDA histogram");
    m.def("sigmoid_cuda", &sigmoid_cuda, "CUDA sigmoid");
    m.def("relu_cuda", &relu_cuda, "CUDA relu");
    m.def("elu_cuda", &elu_cuda, "CUDA ELU");
    m.def("gelu_cuda", &gelu_cuda, "CUDA GELU");
    m.def("swish_cuda", &swish_cuda, "CUDA SWISH");
}
