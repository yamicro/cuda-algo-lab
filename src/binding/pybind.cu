#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <hip/hip_runtime.h>
#include "cuda/add.cu"
#include "cuda/histgram.cu"

void add_cuda(pybind11::array_t<float> a, pybind11::array_t<float> b, pybind11::array_t<float> c) {
    auto buf_a = a.unchecked<1>();
    auto buf_b = b.unchecked<1>();
    auto buf_c = c.mutable_unchecked<1>();
    int n = buf_a.size();

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, n * sizeof(float));
    hipMalloc(&d_b, n * sizeof(float));

    hipMemcpy(d_a, buf_a.data(0), n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, buf_b.data(0), n * sizeof(float), hipMemcpyHostToDevice);

    add_kernel<<<(n+255)/256, 256>>>(d_a, d_b, d_c, n);

    hipMemcpy(buf_c.mutable_data(0), d_c, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}


void histogram_cuda(pybind11::array_t<int> a, pybind11::array_t<int> y) {
    auto buf_a = a.unchecked<1>();
    auto buf_y = y.mutable_unchecked<1>();
    int N = buf_a.size();

    int *d_a, *d_y;
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_y, y.size() * sizeof(int));

    hipMemcpy(d_a, buf_a.data(0), N * sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(d_y, buf_y.data(0), buf_y.size(0) * sizeof(int), hipMemcpyHostToDevice);

    histogram_kernel<<<(N+255)/256, 256>>>(d_a, d_y, N);

    hipMemcpy(buf_y.mutable_data(0), d_y, buf_y.size() * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_y);
}

PYBIND11_MODULE(binding, m) {
    m.def("add_cuda", &add_cuda, "CUDA add two arrays");
    m.def("histogram_cuda", &histogram_cuda, "CUDA histogram");
}
