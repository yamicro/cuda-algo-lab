#include <iostream>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <hip/hip_runtime.h>
#include "bench_cuda/benchmark_cuda.h"
#include "cuda/add.cu"
#include "cuda/histgram.cu"
#include "cuda/sigmod.cu"
#include "cuda/relu.cu"
#include "cuda/elu.cu"
#include "cuda/gelu.cu"
#include "cuda/swish.cu"
#include "cuda/embedding.cu"
#include "cuda/mat_transpose.cu"
#include "cuda/warp_reduce_sum.cu"





float add_cuda(pybind11::array_t<float> a, pybind11::array_t<float> b, pybind11::array_t<float> c) {
    auto buf_a = a.unchecked<1>();
    auto buf_b = b.unchecked<1>();
    auto buf_c = c.mutable_unchecked<1>();
    int n = buf_a.size();

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, n * sizeof(float));
    hipMalloc(&d_b, n * sizeof(float));

    hipMemcpy(d_a, buf_a.data(0), n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, buf_b.data(0), n * sizeof(float), hipMemcpyHostToDevice);

    float elapsed = benchmark_kernel([&]() {
        add_kernel<<<(n+255)/256, 256>>>(d_a, d_b, d_c, n);
    }, 3, 10);


    hipMemcpy(buf_c.mutable_data(0), d_c, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return elapsed;
}


float histogram_cuda(pybind11::array_t<int> a, pybind11::array_t<int> y) {
    auto buf_a = a.unchecked<1>();
    auto buf_y = y.mutable_unchecked<1>();
    int N = buf_a.size();

    int *d_a, *d_y;
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_y, y.size() * sizeof(int));

    hipMemcpy(d_a, buf_a.data(0), N * sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(d_y, buf_y.data(0), buf_y.size(0) * sizeof(int), hipMemcpyHostToDevice);
    float elapsed = benchmark_kernel([&]() {
        histogram_kernel<<<(N+255)/256, 256>>>(d_a, d_y, N);
    }, 3, 10);


    hipMemcpy(buf_y.mutable_data(0), d_y, buf_y.size() * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_y);

    return elapsed;
}

float sigmoid_cuda(pybind11::array_t<float> x, pybind11::array_t<float> y) {
    auto buf_x = x.unchecked<1>();
    auto buf_y = y.mutable_unchecked<1>();
    int N = buf_x.size();

    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    hipMemcpy(d_x, buf_x.data(0), N * sizeof(float), hipMemcpyHostToDevice);

    float elapsed = benchmark_kernel([&]() {
        sigmoid_kernel<<<(N+255)/256, 256>>>(d_x, d_y, N);
    }, 3, 10);

    hipMemcpy(buf_y.mutable_data(0), d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

    return elapsed;
}

float relu_cuda(pybind11::array_t<float> x, pybind11::array_t<float> y) {
    auto buf_x = x.unchecked<1>();
    auto buf_y = y.mutable_unchecked<1>();
    int N = buf_x.size();

    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    hipMemcpy(d_x, buf_x.data(0), N * sizeof(float), hipMemcpyHostToDevice);

    // benchmark 包裹 kernel 启动
    float elapsed = benchmark_kernel([&]() {
        relu_kernel<<<(N+255)/256, 256>>>(d_x, d_y, N);
    },3, 10);

    hipMemcpy(buf_y.mutable_data(0), d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

    return elapsed;
}

float elu_cuda(pybind11::array_t<float> x, pybind11::array_t<float> y) {
    auto buf_x = x.unchecked<1>();
    auto buf_y = y.mutable_unchecked<1>();
    int N = buf_x.size();

    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    hipMemcpy(d_x, buf_x.data(0), N * sizeof(float), hipMemcpyHostToDevice);

	float elapsed = benchmark_kernel([&]() {
        relu_kernel<<<(N+255)/256, 256>>>(d_x, d_y, N);
    },3, 10);

    hipMemcpy(buf_y.mutable_data(0), d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    return elapsed;
}

float gelu_cuda(pybind11::array_t<float> x, pybind11::array_t<float> y) {
    auto buf_x = x.unchecked<1>();
    auto buf_y = y.mutable_unchecked<1>();
    int N = buf_x.size();

    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    hipMemcpy(d_x, buf_x.data(0), N * sizeof(float), hipMemcpyHostToDevice);

    float elapsed = benchmark_kernel([&]() {
    	gelu_kernel<<<(N + 255) / 256, 256>>>(d_x, d_y, N);
    },3, 10);

    hipMemcpy(buf_y.mutable_data(0), d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    return elapsed;
}

float swish_cuda(pybind11::array_t<float> x, pybind11::array_t<float> y) {
    auto buf_x = x.unchecked<1>();
    auto buf_y = y.mutable_unchecked<1>();
    int N = buf_x.size();

    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    hipMemcpy(d_x, buf_x.data(0), N * sizeof(float), hipMemcpyHostToDevice);

    float elapsed = benchmark_kernel([&]() {
    	swish_f32_kernel<<<(N + 255) / 256, 256>>>(d_x, d_y, N);
    },3, 10);

    hipMemcpy(buf_y.mutable_data(0), d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    return elapsed;
}

float embedding_cuda(pybind11::array_t<int> input,pybind11::array_t<float> weights, pybind11::array_t<float> output) {
    auto buf_input = input.unchecked<1>();
    auto buf_weights = weights.unchecked<2>();
    auto buf_output_info = output.request();
    float *h_output = static_cast<float *>(buf_output_info.ptr);

    int N = buf_input.size();
    int D = buf_weights.shape(1);

    int *d_indices;
    float *d_weights, *d_output;
    hipMalloc(&d_indices, N * sizeof(int));
    hipMalloc(&d_weights, buf_weights.shape(0) * D * sizeof(float));
    hipMalloc(&d_output, N * D * sizeof(float));

    hipMemcpy(d_indices, buf_input.data(0), N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_weights, buf_weights.data(0, 0), buf_weights.shape(0) * D * sizeof(float), hipMemcpyHostToDevice);

    float elapsed = benchmark_kernel([&]() {
        embedding_f32_kernel<<<(N + 255) / 256, 256>>>(d_indices, d_weights, d_output, N, D);
    }, 3, 10);

    hipMemcpy(h_output, d_output, N * D * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_indices);
    hipFree(d_weights);
    hipFree(d_output);

    return elapsed;
}

float mat_transpose_cuda(pybind11::array_t<int> input, pybind11::array_t<float> output) {
    auto buf_input = input.unchecked<2>();
    auto buf_output = output.mutable_unchecked<2>();

    int N = buf_input.shape(0);
    int D = buf_input.shape(1);

    float* d_input;
    float* d_output;
    hipMalloc(&d_input, N * D * sizeof(float));
    hipMalloc(&d_output, N * D * sizeof(float));

    hipMemcpy(d_input, buf_input.data(0, 0), N * D * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_output, buf_input.data(0, 0), N * D * sizeof(float), hipMemcpyHostToDevice);


    float elapsed = benchmark_kernel([&]() {
        mat_transpose_f32_col2row_kernel<<<(N + 255) / 256, 256>>>(d_input, d_output, N, D);
    }, 3, 10);

    hipMemcpy(buf_output.mutable_data(0,0), d_output, N * D * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return elapsed;
}


float warp_reduce_sum_cuda(pybind11::array_t<float> input, pybind11::array_t<float> output) {
    auto buf_in = input.unchecked<1>();
    auto buf_out = output.mutable_unchecked<1>();

    int N = buf_in.size();

    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, (N / 32) * sizeof(float));

    hipMemcpy(d_x, buf_in.data(0), N * sizeof(float), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    float elapsed = benchmark_kernel([&]() {
    	block_all_reduce_sum_f32_f32_kernel<<<blocks, threads>>>(d_x, d_y, N);
    }, 3, 10);


    hipMemcpy(buf_out.mutable_data(0), d_y, (N / 32) * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    return elapsed;
}


PYBIND11_MODULE(binding, m) {
    m.def("add_cuda", &add_cuda, "CUDA add two arrays");
    m.def("histogram_cuda", &histogram_cuda, "CUDA histogram");
    m.def("sigmoid_cuda", &sigmoid_cuda, "CUDA sigmoid");
    m.def("relu_cuda", &relu_cuda, "CUDA relu");
    m.def("elu_cuda", &elu_cuda, "CUDA ELU");
    m.def("gelu_cuda", &gelu_cuda, "CUDA GELU");
    m.def("swish_cuda", &swish_cuda, "CUDA SWISH");
    m.def("embedding_cuda", &embedding_cuda, "CUDA embedding");
    m.def("mat_transpose_cuda", &mat_transpose_cuda, "CUDA mat_transpose transpose");
	m.def("warp_reduce_sum_cuda", &warp_reduce_sum_cuda, "CUDA warp reduce sum");
}
