#include "hip/hip_runtime.h"
#include <iostream>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include "util/benchmark_cuda.h"
#include "cuda/add.cu"
#include "cuda/histgram.cu"
#include "cuda/sigmod.cu"
#include "cuda/relu.cu"
#include "cuda/elu.cu"
#include "cuda/gelu.cu"
#include "cuda/swish.cu"
#include "cuda/embedding.cu"
#include "cuda/mat_transpose.cu"
#include "cuda/warp_reduce_sum.cu"

#ifndef PYBIND11_HALF_T_DEFINED
#define PYBIND11_HALF_T_DEFINED
namespace pybind11 { using half_t = uint16_t; }
#endif




float add_cuda(pybind11::array_t<float> a, pybind11::array_t<float> b, pybind11::array_t<float> c) {
    auto buf_a = a.unchecked<1>();
    auto buf_b = b.unchecked<1>();
    auto buf_c = c.mutable_unchecked<1>();
    int n = buf_a.size();

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, n * sizeof(float));
    hipMalloc(&d_b, n * sizeof(float));

    hipMemcpy(d_a, buf_a.data(0), n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, buf_b.data(0), n * sizeof(float), hipMemcpyHostToDevice);

    float elapsed = benchmark_kernel([&]() {
        add_kernel<<<(n+255)/256, 256>>>(d_a, d_b, d_c, n);
    }, 3, 10);


    hipMemcpy(buf_c.mutable_data(0), d_c, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return elapsed;
}

float add_fp16_pack_cuda_trans_after(pybind11::array_t<float> a, pybind11::array_t<float> b, pybind11::array_t<float> c) {
    auto buf_a = a.unchecked<1>();
    auto buf_b = b.unchecked<1>();
    auto buf_c = c.mutable_unchecked<1>();
    int n = buf_a.size();

    int N = buf_a.size();
    if (N % 8 != 0) throw std::runtime_error("Input size must be divisible by 8.");

    float* h_a = const_cast<float*>(buf_a.data(0));
    half* d_a;

    float* h_b = const_cast<float*>(buf_b.data(0));
    half* d_b;

    hipMalloc(&d_a, N * sizeof(half));
    hipMalloc(&d_b, N * sizeof(half));

    hipMemcpy(d_a, buf_a.data(0), n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, buf_b.data(0), n * sizeof(float), hipMemcpyHostToDevice);
    float* d_c;
    hipMalloc(&d_c, N * sizeof(float));
    hipMemset(d_c, 0, N * sizeof(float));

    float elapsed = benchmark_kernel([&]() {
        add_fp16_pack_kernel<<<(n+255)/256, 256>>>(d_a, d_b, d_c, n);
    }, 3, 10);


    hipMemcpy(buf_c.mutable_data(0), d_c, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return elapsed;
}

float add_fp16_pack_cuda(pybind11::array_t<pybind11::half_t,
                                                  pybind11::array::c_style |
                                                  pybind11::array::forcecast> a,
                                      pybind11::array_t<pybind11::half_t,
                                                  pybind11::array::c_style |
                                                  pybind11::array::forcecast> b) {

    const int N = static_cast<int>(a.size());

    auto out = pybind11::array_t<float>(N);
    float* h_out = out.mutable_data();

    const __half* h_a = reinterpret_cast<const __half*>(a.data());
    const __half* h_b = reinterpret_cast<const __half*>(b.data());

    __half* d_a;  __half* d_b;
    float * d_c;
    hipMalloc(&d_a, N * sizeof(__half));
    hipMalloc(&d_b, N * sizeof(__half));
    hipMalloc(&d_c, N * sizeof(float));

    hipMemcpy(d_a, h_a, N * sizeof(__half), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(__half), hipMemcpyHostToDevice);

    constexpr int THREADS = 128;
    constexpr int VEC = 8;
    int elems_per_block = THREADS * VEC;
    int grid = (N + elems_per_block - 1) / elems_per_block;

    float elapsed = benchmark_kernel([&]() {
        add_fp16_pack_kernel<<<grid, THREADS>>>(d_a, d_b, d_c, N);
    }, 3, 10);

    hipMemcpy(h_out, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return elapsed;
}


float histogram_cuda(pybind11::array_t<int> a, pybind11::array_t<int> y) {
    auto buf_a = a.unchecked<1>();
    auto buf_y = y.mutable_unchecked<1>();
    int N = buf_a.size();

    int *d_a, *d_y;
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_y, y.size() * sizeof(int));

    hipMemcpy(d_a, buf_a.data(0), N * sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(d_y, buf_y.data(0), buf_y.size(0) * sizeof(int), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;
    float elapsed = benchmark_kernel([&]() {
        histogram_kernel<<<blocks, threads>>>(d_a, d_y, N);
    }, 3, 10);


    hipMemcpy(buf_y.mutable_data(0), d_y, buf_y.size() * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_y);

    return elapsed;
}

float sigmoid_cuda(pybind11::array_t<float> x, pybind11::array_t<float> y) {
    auto buf_x = x.unchecked<1>();
    auto buf_y = y.mutable_unchecked<1>();
    int N = buf_x.size();

    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    hipMemcpy(d_x, buf_x.data(0), N * sizeof(float), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    float elapsed = benchmark_kernel([&]() {
        sigmoid_kernel<<<blocks, threads>>>(d_x, d_y, N);
    }, 3, 10);

    hipMemcpy(buf_y.mutable_data(0), d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

    return elapsed;
}

float sigmoid_cuda_fp16_pack(pybind11::array_t<pybind11::half_t,
                                         pybind11::array::c_style |
                                         pybind11::array::forcecast> x,
                             pybind11::array_t<pybind11::half_t,
                                         pybind11::array::c_style |
                                         pybind11::array::forcecast> y,
                             int M) {

    const int N = static_cast<int>(x.size());

    const __half* h_x = reinterpret_cast<const __half*>(x.data());
    __half*       h_y = reinterpret_cast<__half*>(y.mutable_data());

    __half *d_x = nullptr, *d_y = nullptr;
    hipMalloc(&d_x, N * sizeof(__half));
    hipMalloc(&d_y, N * sizeof(__half));

    hipMemcpy(d_x, h_x, N * sizeof(__half), hipMemcpyHostToDevice);

    constexpr int THREADS = 256;
    int blocks = (N + THREADS - 1) / THREADS;

    float elapsed = benchmark_kernel([&](){
        sigmoid_f16x8_pack_kernel<<<blocks, THREADS>>>(d_x, d_y, N);
    }, 3, 10);

    hipMemcpy(h_y, d_y, N * sizeof(__half), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    return elapsed;
}

float relu_cuda(pybind11::array_t<float> x, pybind11::array_t<float> y) {
    auto buf_x = x.unchecked<1>();
    auto buf_y = y.mutable_unchecked<1>();
    int N = buf_x.size();

    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    hipMemcpy(d_x, buf_x.data(0), N * sizeof(float), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    float elapsed = benchmark_kernel([&]() {
        relu_kernel<<<blocks, threads>>>(d_x, d_y, N);
    },3, 10);

    hipMemcpy(buf_y.mutable_data(0), d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

    return elapsed;
}

float relu_cuda_fp16_pack(pybind11::array_t<pybind11::half_t,
                                         pybind11::array::c_style |
                                         pybind11::array::forcecast> x,
                             pybind11::array_t<pybind11::half_t,
                                         pybind11::array::c_style |
                                         pybind11::array::forcecast> y) {

    const int N = static_cast<int>(x.size());

    const __half* h_x = reinterpret_cast<const __half*>(x.data());
    __half*       h_y = reinterpret_cast<__half*>(y.mutable_data());

    __half *d_x = nullptr, *d_y = nullptr;
    hipMalloc(&d_x, N * sizeof(__half));
    hipMalloc(&d_y, N * sizeof(__half));

    hipMemcpy(d_x, h_x, N * sizeof(__half), hipMemcpyHostToDevice);

    constexpr int THREADS = 256;
    int blocks = (N + THREADS - 1) / THREADS;

    float elapsed = benchmark_kernel([&](){
        relu_f16x8_pack_kernel<<<blocks, THREADS>>>(d_x, d_y, N);
    }, 3, 10);

    hipMemcpy(h_y, d_y, N * sizeof(__half), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    return elapsed;
}

float elu_cuda(pybind11::array_t<float> x, pybind11::array_t<float> y) {
    auto buf_x = x.unchecked<1>();
    auto buf_y = y.mutable_unchecked<1>();
    int N = buf_x.size();

    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    hipMemcpy(d_x, buf_x.data(0), N * sizeof(float), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;

	float elapsed = benchmark_kernel([&]() {
        elu_f32_kernel<<<blocks, threads>>>(d_x, d_y, N);
    },3, 10);

    hipMemcpy(buf_y.mutable_data(0), d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    return elapsed;
}

float elu_cuda_fp16_pack(pybind11::array_t<pybind11::half_t,
                                         pybind11::array::c_style |
                                         pybind11::array::forcecast> x) {

    const int N = static_cast<int>(x.size());

    auto y = pybind11::array_t<float>(N);
    float* h_out = y.mutable_data();

    const half* h_in = reinterpret_cast<const half*>(x.data());
    half*  d_in  = nullptr;
    float* d_out = nullptr;
    hipMalloc(&d_in,  N * sizeof(half));
    hipMalloc(&d_out, N * sizeof(float));

    hipMemcpy(d_in, h_in, N * sizeof(half), hipMemcpyHostToDevice);

    constexpr int THREADS = 128;
    constexpr int VEC = 8;
    int elems_per_block = THREADS * VEC;
    int grid = (N + elems_per_block - 1) / elems_per_block;

    auto t0 = std::chrono::high_resolution_clock::now();
    float elapsed = benchmark_kernel([&]() {
        elu_f16x8_pack_kernel<<<grid, THREADS>>>(d_in, d_out, N);
    },3, 10);
    hipDeviceSynchronize();
    auto t1 = std::chrono::high_resolution_clock::now();

    /* ---- D→H ---- */
    hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);

    float ms = std::chrono::duration<float, std::milli>(t1 - t0).count();
    return elapsed;
}

float gelu_cuda(pybind11::array_t<float> x, pybind11::array_t<float> y) {
    auto buf_x = x.unchecked<1>();
    auto buf_y = y.mutable_unchecked<1>();
    int N = buf_x.size();

    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    hipMemcpy(d_x, buf_x.data(0), N * sizeof(float), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    float elapsed = benchmark_kernel([&]() {
    	gelu_kernel<<<blocks, threads>>>(d_x, d_y, N);
    },3, 10);

    hipMemcpy(buf_y.mutable_data(0), d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    return elapsed;
}

float gelu_cuda_fp16_pack(pybind11::array_t<pybind11::half_t,
                                      pybind11::array::c_style |
                                      pybind11::array::forcecast> x,
                          pybind11::array_t<pybind11::half_t,
                                      pybind11::array::c_style |
                                      pybind11::array::forcecast> y) {
    const int N = static_cast<int>(x.size());

    const __half* h_x = reinterpret_cast<const __half*>(x.data());
    __half*       h_y = reinterpret_cast<__half*>(y.mutable_data());

    __half *d_x = nullptr, *d_y = nullptr;
    hipMalloc(&d_x, N * sizeof(__half));
    hipMalloc(&d_y, N * sizeof(__half));

    hipMemcpy(d_x, h_x, N * sizeof(__half), hipMemcpyHostToDevice);

    constexpr int THREADS = 256;
    int blocks = (N + THREADS - 1) / THREADS;

    float elapsed = benchmark_kernel([&]() {
        gelu_f16x8_pack_kernel<<<blocks, THREADS>>>(d_x, d_y, N);
    }, 3, 10);

    hipMemcpy(h_y, d_y, N * sizeof(__half), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    return elapsed;
}

float swish_cuda(pybind11::array_t<float> x, pybind11::array_t<float> y) {
    auto buf_x = x.unchecked<1>();
    auto buf_y = y.mutable_unchecked<1>();
    int N = buf_x.size();

    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    hipMemcpy(d_x, buf_x.data(0), N * sizeof(float), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    float elapsed = benchmark_kernel([&]() {
    	swish_f32_kernel<<<blocks, threads>>>(d_x, d_y, N);
    },3, 10);

    hipMemcpy(buf_y.mutable_data(0), d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    return elapsed;
}

float swish_cuda_fp16_pack(pybind11::array_t<pybind11::half_t,
                                      pybind11::array::c_style |
                                      pybind11::array::forcecast> x,
                          pybind11::array_t<pybind11::half_t,
                                      pybind11::array::c_style |
                                      pybind11::array::forcecast> y) {
    const int N = static_cast<int>(x.size());

    const __half* h_x = reinterpret_cast<const __half*>(x.data());
    __half*       h_y = reinterpret_cast<__half*>(y.mutable_data());

    __half *d_x = nullptr, *d_y = nullptr;
    hipMalloc(&d_x, N * sizeof(__half));
    hipMalloc(&d_y, N * sizeof(__half));

    hipMemcpy(d_x, h_x, N * sizeof(__half), hipMemcpyHostToDevice);

    constexpr int THREADS = 256;
    int blocks = (N + THREADS - 1) / THREADS;

    float elapsed = benchmark_kernel([&]() {
        swish_f16x8_pack_kernel<<<blocks, THREADS>>>(d_x, d_y, N);
    }, 3, 10);

    /* ---------- D→H ---------- */
    hipMemcpy(h_y, d_y, N * sizeof(__half), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    return elapsed;
}

float embedding_cuda(pybind11::array_t<int> input,pybind11::array_t<float> weights, pybind11::array_t<float> output) {
    auto buf_input = input.unchecked<1>();
    auto buf_weights = weights.unchecked<2>();
    auto buf_output_info = output.request();
    float *h_output = static_cast<float *>(buf_output_info.ptr);

    int N = buf_input.size();
    int D = buf_weights.shape(1);

    int *d_indices;
    float *d_weights, *d_output;
    hipMalloc(&d_indices, N * sizeof(int));
    hipMalloc(&d_weights, buf_weights.shape(0) * D * sizeof(float));
    hipMalloc(&d_output, N * D * sizeof(float));

    hipMemcpy(d_indices, buf_input.data(0), N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_weights, buf_weights.data(0, 0), buf_weights.shape(0) * D * sizeof(float), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    float elapsed = benchmark_kernel([&]() {
        embedding_f32_kernel<<<blocks, threads>>>(d_indices, d_weights, d_output, N, D);
    }, 3, 10);

    hipMemcpy(h_output, d_output, N * D * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_indices);
    hipFree(d_weights);
    hipFree(d_output);

    return elapsed;
}

float embedding_cuda_fp16_pack(pybind11::array_t<int,
                                      pybind11::array::c_style |
                                      pybind11::array::forcecast>      input,
                          pybind11::array_t<pybind11::half_t,
                                      pybind11::array::c_style |
                                      pybind11::array::forcecast>      weights,
                          pybind11::array_t<pybind11::half_t,
                                      pybind11::array::c_style |
                                      pybind11::array::forcecast>      output) {
    auto in  = input.unchecked<1>();
    auto w   = weights.unchecked<2>();
    auto out_info = output.request();
    __half* h_out = reinterpret_cast<__half*>(out_info.ptr);

    const int N = static_cast<int>(in.size());
    const int D = static_cast<int>(w.shape(1));
    const int V = static_cast<int>(w.shape(0));
    if (output.size() != static_cast<size_t>(N * D))
        throw std::runtime_error("output shape 不匹配");

    int*   d_idx     = nullptr;
    __half* d_w      = nullptr;
    __half* d_out    = nullptr;
    hipMalloc(&d_idx,  N * sizeof(int));
    hipMalloc(&d_w,   V * D * sizeof(__half));
    hipMalloc(&d_out, N * D * sizeof(__half));

    const int*   h_idx = in.data(0);
    const __half* h_w  = reinterpret_cast<const __half*>(w.data(0,0));

    hipMemcpy(d_idx, h_idx, N * sizeof(int),            hipMemcpyHostToDevice);
    hipMemcpy(d_w,   h_w,  V * D * sizeof(__half),      hipMemcpyHostToDevice);

    int threads = 256;
    int blocks  = (N + threads - 1) / threads;

    float elapsed = benchmark_kernel([&](){
        embedding_f16x8_pack_kernel<<<blocks, threads>>>(d_idx, d_w, d_out, N, D);
    }, 3, 10);

    hipMemcpy(h_out, d_out, N * D * sizeof(__half),     hipMemcpyDeviceToHost);

    hipFree(d_idx);
    hipFree(d_w);
    hipFree(d_out);

    return elapsed;
}

float mat_transpose_cuda(pybind11::array_t<int> input, pybind11::array_t<float> output) {
    auto buf_input = input.unchecked<2>();
    auto buf_output = output.mutable_unchecked<2>();

    int N = buf_input.shape(0);
    int D = buf_input.shape(1);

    float* d_input;
    float* d_output;
    hipMalloc(&d_input, N * D * sizeof(float));
    hipMalloc(&d_output, N * D * sizeof(float));

    hipMemcpy(d_input, buf_input.data(0, 0), N * D * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_output, buf_input.data(0, 0), N * D * sizeof(float), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    float elapsed = benchmark_kernel([&]() {
        mat_transpose_f32_col2row_kernel<<<blocks, threads>>>(d_input, d_output, N, D);
    }, 3, 10);

    hipMemcpy(buf_output.mutable_data(0,0), d_output, N * D * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return elapsed;
}


float warp_reduce_sum_cuda(pybind11::array_t<float> input, pybind11::array_t<float> output) {
    auto buf_in = input.unchecked<1>();
    auto buf_out = output.mutable_unchecked<1>();

    int N = buf_in.size();

    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, (N / 32) * sizeof(float));

    hipMemcpy(d_x, buf_in.data(0), N * sizeof(float), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    float elapsed = benchmark_kernel([&]() {
    	block_all_reduce_sum_f32_f32_kernel<<<blocks, threads>>>(d_x, d_y, N);
    }, 3, 10);


    hipMemcpy(buf_out.mutable_data(0), d_y, (N / 32) * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    return elapsed;
}

float warp_reduce_fp16_cuda_trans_after(pybind11::array_t<float> input, pybind11::array_t<float> output) {
    auto buf_in = input.unchecked<1>();
    auto buf_out = output.mutable_unchecked<1>();

    int N = buf_in.size();
    if (N % 8 != 0) throw std::runtime_error("Input size must be divisible by 8.");

    float* h_input = const_cast<float*>(buf_in.data(0));
    half* d_input;
    float* d_output;

    hipMalloc(&d_input, N * sizeof(half));
    hipMalloc(&d_output, sizeof(float));
    hipMemset(d_output, 0, sizeof(float));

    // convert to half
    std::vector<half> h_half(N);
    for (int i = 0; i < N; ++i)
        h_half[i] = __float2half(h_input[i]);

    hipMemcpy(d_input, h_half.data(), N * sizeof(half), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    float elapsed = benchmark_kernel([&]() {
    	block_all_reduce_sum_f16x8_pack_f16_kernel<<<blocks, threads>>>(d_input, d_output, N);
    }, 3, 10);

    hipMemcpy(buf_out.mutable_data(0), d_output, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    return elapsed;
}

float warp_reduce_fp16_cuda(pybind11::array_t<pybind11::half_t,
                                       pybind11::array::c_style |
                                       pybind11::array::forcecast> input,
                            pybind11::array_t<float,
                                       pybind11::array::c_style | pybind11::array::forcecast> output) {

    const int N = static_cast<int>(input.size());

    const __half* h_in = reinterpret_cast<const __half*>(input.data());
    float*        h_out = output.mutable_data();

    __half* d_in  = nullptr;
    float*  d_out = nullptr;
    hipMalloc(&d_in,  N * sizeof(__half));
    hipMalloc(&d_out, sizeof(float));
    hipMemset(d_out, 0, sizeof(float));

    hipMemcpy(d_in, h_in, N * sizeof(__half), hipMemcpyHostToDevice);

    constexpr int threads = 256;
    int blocks = (N + threads - 1) / threads;
    float elapsed = benchmark_kernel([&]() {
    	block_all_reduce_sum_f16x8_pack_f16_kernel<<<blocks, threads>>>(d_in, d_out, N);
    }, 3, 10);
    hipMemcpy(h_out, d_out, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
    return elapsed;
}



PYBIND11_MODULE(binding, m) {
    m.def("add_cuda", &add_cuda, "CUDA add two arrays");
    m.def("add_fp16_pack_cuda", &add_fp16_pack_cuda, "CUDA add two arrays in fp16");
    m.def("histogram_cuda", &histogram_cuda, "CUDA histogram");
    m.def("sigmoid_cuda", &sigmoid_cuda, "CUDA sigmoid");
    m.def("sigmoid_cuda_fp16_pack", &sigmoid_cuda_fp16_pack, "CUDA sigmoid fp16");
    m.def("relu_cuda", &relu_cuda, "CUDA relu");
    m.def("relu_cuda_fp16_pack", &relu_cuda_fp16_pack, "CUDA relu fp16");
    m.def("elu_cuda", &elu_cuda, "CUDA ELU");
    m.def("elu_cuda_fp16_pack", &elu_cuda_fp16_pack, "CUDA elu_cuda_fp16_pack");
    m.def("gelu_cuda", &gelu_cuda, "CUDA GELU");
    m.def("gelu_cuda_fp16_pack", &gelu_cuda_fp16_pack, "CUDA GELU gelu_cuda_fp16_pack");
    m.def("swish_cuda", &swish_cuda, "CUDA SWISH");
    m.def("swish_cuda_fp16_pack", &swish_cuda_fp16_pack, "CUDA fp16 swish");
    m.def("embedding_cuda", &embedding_cuda, "CUDA embedding");
    m.def("embedding_cuda_fp16_pack", &embedding_cuda_fp16_pack, "CUDA embedding_cuda_fp16_pack");
    m.def("mat_transpose_cuda", &mat_transpose_cuda, "CUDA mat_transpose transpose");
	m.def("warp_reduce_sum_cuda", &warp_reduce_sum_cuda, "CUDA warp reduce sum");
    m.def("warp_reduce_fp16_cuda", &warp_reduce_fp16_cuda, "warp reduce fp16");
}
