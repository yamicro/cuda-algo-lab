
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <hip/hip_runtime.h>
#include "cuda/add.cu"

void add_cuda(pybind11::array_t<float> a, pybind11::array_t<float> b, pybind11::array_t<float> c) {
    auto buf_a = a.unchecked<1>();
    auto buf_b = b.unchecked<1>();
    auto buf_c = c.mutable_unchecked<1>();
    int n = buf_a.size();

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, n * sizeof(float));
    hipMalloc(&d_b, n * sizeof(float));
    hipMalloc(&d_c, n * sizeof(float));

    hipMemcpy(d_a, buf_a.data(0), n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, buf_b.data(0), n * sizeof(float), hipMemcpyHostToDevice);

    add_kernel<<<(n+255)/256, 256>>>(d_a, d_b, d_c, n);

    hipMemcpy(buf_c.mutable_data(0), d_c, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}

PYBIND11_MODULE(binding, m) {
    m.def("add_cuda", &add_cuda, "CUDA add two arrays");
}
