#include "hip/hip_runtime.h"
//
// Created by yami on 25-4-15.
//
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdio.h>
#include <stdlib.h>
#include "util/half_exp.h"

#define MAX_EXP_F32  88.3762626647949f
#define MIN_EXP_F32 -88.3762626647949f
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])
#define MAX_EXP_F16 __float2half(11.089866488461016f)
#define MIN_EXP_F16 __float2half(-9.704060527839234f)

__global__ void sigmoid_kernel(float* x, float* y, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    float v = x[idx];
    v = fminf(fmaxf(v, MIN_EXP_F32), MAX_EXP_F32);
    y[idx] = 1.0f / (1.0f + expf(-v));
  }
}

__global__ void sigmoid_f16x8_pack_kernel(half *x, half *y, int N) {
  int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 8;
  const half f = __float2half(1.0f);
  // temporary register(memory), .local space in ptx, addressable
  half pack_x[8], pack_y[8]; // 8x16 bits=128 bits.
  // reinterpret as float4 and load 128 bits in 1 memory issue.
  LDST128BITS(pack_x[0]) = LDST128BITS(x[idx]); // load 128 bits

#pragma unroll
  for (int i = 0; i < 8; ++i) {
    half v = __hmin(__hmax(pack_x[i], MIN_EXP_F16), MAX_EXP_F16);
    pack_y[i] = f / (f + half_exp(-v));
  }
  // reinterpret as float4 and store 128 bits in 1 memory issue.
  if ((idx + 7) < N) {
    LDST128BITS(y[idx]) = LDST128BITS(pack_y[0]);
  }
}