//
// Created by yami on 25-4-16.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#define HALF2(value) (reinterpret_cast<half2 *>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])
__global__ void relu_kernel(float* x, float* y, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) { y[idx] = fmaxf(0.0f, x[idx]); }
}

__global__ void relu_f16x8_pack_kernel(half *x, half *y, int N) {
  int idx = 8 * (blockIdx.x * blockDim.x + threadIdx.x);
  const half2 z2 = {__float2half(0.0f), __float2half(0.0f)};
  // temporary register(memory), .local space in ptx, addressable
  half pack_x[8], pack_y[8]; // 8x16 bits=128 bits.
  // reinterpret as float4 and load 128 bits in 1 memory issue.
  LDST128BITS(pack_x[0]) = LDST128BITS(x[idx]); // load 128 bits

#pragma unroll
  for (int i = 0; i < 8; i += 2) {
    // __hmax2 for half2 x 4
    HALF2(pack_y[i]) = __hmax2(HALF2(pack_x[i]), z2);
  }
  // reinterpret as float4 and store 128 bits in 1 memory issue.
  if ((idx + 7) < N) {
    LDST128BITS(y[idx]) = LDST128BITS(pack_y[0]);
  }
}