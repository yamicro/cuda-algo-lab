#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])

__global__ void add_kernel(const float* a, const float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) c[idx] = a[idx] + b[idx];
}

__global__ void add_fp16_pack_kernel(half* a,
                                     half* b,
                                     float*       c,
                                     int          n) {
    int base = (blockIdx.x * blockDim.x + threadIdx.x) * 8;
    if (base >= n) return;                     // 超界直接退出

    /* ---- 本地寄存器缓存 ---- */
    half pack_a[8];
    half pack_b[8];

    LDST128BITS(pack_a[0]) = LDST128BITS(a[base]);
    LDST128BITS(pack_b[0]) = LDST128BITS(b[base]);

    /* ---- 逐元素 half 运算 → fp32 写回 ---- */
#pragma unroll
    for (int i = 0; i < 8; ++i) {
        int idx = base + i;
        if (idx < n) {                         // 末块可能不足 8 个元素
            half  sum_h = __hadd(pack_a[i], pack_b[i]);   // FP16 加法
            c[idx]      = __half2float(sum_h);            // 写回 FP32
        }
    }
}